#include "hip/hip_runtime.h"
#include "vector_add.cuh"

__global__ void vecAddKernel(const float* A, const float* B, float* C, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void vecAddGpu(const float* A, const float* B, float* C, int n) {
    float *d_A, *d_B, *d_C;
    int size = n * sizeof(float);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int nblocks = (n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;
    vecAddKernel<<<nblocks, THREAD_PER_BLOCK>>> (d_A, d_B, d_C, n);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}